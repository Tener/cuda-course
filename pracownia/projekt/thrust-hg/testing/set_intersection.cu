#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

template<typename Vector>
void TestSetIntersectionSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(2);
  ref[0] = 0; ref[1] = 4;

  Vector result(2);

  Iterator end = thrust::set_intersection(a.begin(), a.end(),
                                          b.begin(), b.end(),
                                          result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetIntersectionSimple);


template<typename T>
void TestSetIntersection(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);
  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersection);


template<typename T>
void TestSetIntersectionEquivalentRanges(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_a = temp; thrust::sort(h_a.begin(), h_a.end());
  thrust::host_vector<T> h_b = h_a;

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionEquivalentRanges);


template<typename T>
void TestSetIntersectionMultiset(const size_t n)
{
  thrust::host_vector<T> temp = unittest::random_integers<T>(2 * n);

  // restrict elements to [min,13)
  for(typename thrust::host_vector<T>::iterator i = temp.begin();
      i != temp.end();
      ++i)
  {
    int temp = static_cast<int>(*i);
    temp %= 13;
    *i = temp;
  }

  thrust::host_vector<T> h_a(temp.begin(), temp.begin() + n);
  thrust::host_vector<T> h_b(temp.begin() + n, temp.end());

  thrust::sort(h_a.begin(), h_a.end());
  thrust::sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionMultiset);


template<typename U>
  void TestSetIntersectionKeyValue(size_t n)
{
  typedef key_value<U,U> T;

  thrust::host_vector<U> h_keys_a   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_a = unittest::random_integers<U>(n);

  thrust::host_vector<U> h_keys_b   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_b = unittest::random_integers<U>(n);

  thrust::host_vector<T> h_a(n), h_b(n);
  for(size_t i = 0; i < n; ++i)
  {
    h_a[i] = T(h_keys_a[i], h_values_a[i]);
    h_b[i] = T(h_keys_b[i], h_values_b[i]);
  }

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(n);
  thrust::device_vector<T> d_result(n);

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_intersection(h_a.begin(), h_a.end(),
                                   h_b.begin(), h_b.end(),
                                   h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_intersection(d_a.begin(), d_a.end(),
                                   d_b.begin(), d_b.end(),
                                   d_result.begin());

  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL_QUIET(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetIntersectionKeyValue);

