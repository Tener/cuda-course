#include <unittest/unittest.h>
#include <thrust/set_operations.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

template<typename Vector>
void TestSetUnionSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(4);

  a[0] = 0; a[1] = 2; a[2] = 4;
  b[0] = 0; b[1] = 3; b[2] = 3; b[3] = 4;

  Vector ref(5);
  ref[0] = 0; ref[1] = 2; ref[2] = 3; ref[3] = 3; ref[4] = 4;

  Vector result(5);

  Iterator end = thrust::set_union(a.begin(), a.end(),
                                   b.begin(), b.end(),
                                   result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetUnionSimple);


template<typename Vector>
void TestSetUnionWithEquivalentElementsSimple(void)
{
  typedef typename Vector::iterator Iterator;

  Vector a(3), b(5);

  a[0] = 0; a[1] = 2; a[2] = 2;
  b[0] = 0; b[1] = 2; b[2] = 2; b[3] = 2; b[4] = 3;

  Vector ref(5);
  ref[0] = 0; ref[1] = 2; ref[2] = 2; ref[3] = 2; ref[4] = 3;

  Vector result(5);

  Iterator end = thrust::set_union(a.begin(), a.end(),
                                   b.begin(), b.end(),
                                   result.begin());

  ASSERT_EQUAL_QUIET(result.end(), end);
  ASSERT_EQUAL(ref, result);
}
DECLARE_VECTOR_UNITTEST(TestSetUnionWithEquivalentElementsSimple);


template<typename T>
void TestSetUnion(const size_t n)
{
  thrust::host_vector<T> h_a = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_b = unittest::random_integers<T>(n);

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(d_a.size() + d_b.size());

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_union(h_a.begin(), h_a.end(),
                            h_b.begin(), h_b.end(),
                            h_result.begin());
  h_result.resize(h_end - h_result.begin());

  d_end = thrust::set_union(d_a.begin(), d_a.end(),
                            d_b.begin(), d_b.end(),
                            d_result.begin());
  d_result.resize(d_end - d_result.begin());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnion);


template<typename U>
  void TestSetUnionKeyValue(size_t n)
{
  typedef key_value<U,U> T;

  thrust::host_vector<U> h_keys_a   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_a = unittest::random_integers<U>(n);

  thrust::host_vector<U> h_keys_b   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_b = unittest::random_integers<U>(n);

  thrust::host_vector<T> h_a(n), h_b(n);
  for(size_t i = 0; i < n; ++i)
  {
    h_a[i] = T(h_keys_a[i], h_values_a[i]);
    h_b[i] = T(h_keys_b[i], h_values_b[i]);
  }

  thrust::stable_sort(h_a.begin(), h_a.end());
  thrust::stable_sort(h_b.begin(), h_b.end());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(d_a.size() + d_b.size());

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_union(h_a.begin(), h_a.end(),
                            h_b.begin(), h_b.end(),
                            h_result.begin());
  h_result.erase(h_end, h_result.end());

  d_end = thrust::set_union(d_a.begin(), d_a.end(),
                            d_b.begin(), d_b.end(),
                            d_result.begin());
  d_result.erase(d_end, d_result.end());

  ASSERT_EQUAL_QUIET(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionKeyValue);


template<typename T>
  void TestSetUnionDescending(size_t n)
{
  thrust::host_vector<T> h_a = unittest::random_integers<T>(n);
  thrust::host_vector<T> h_b = unittest::random_integers<T>(n);

  thrust::stable_sort(h_a.begin(), h_a.end(), thrust::greater<T>());
  thrust::stable_sort(h_b.begin(), h_b.end(), thrust::greater<T>());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T> h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(d_a.size() + d_b.size());

  typename thrust::host_vector<T>::iterator h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_union(h_a.begin(), h_a.end(),
                            h_b.begin(), h_b.end(),
                            h_result.begin(),
                            thrust::greater<T>());
  h_result.erase(h_end, h_result.end());

  d_end = thrust::set_union(d_a.begin(), d_a.end(),
                            d_b.begin(), d_b.end(),
                            d_result.begin(),
                            thrust::greater<T>());
  d_result.erase(d_end, d_result.end());

  ASSERT_EQUAL(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionDescending);


template<typename U>
  void TestSetUnionKeyValueDescending(size_t n)
{
  typedef key_value<U,U> T;

  thrust::host_vector<U> h_keys_a   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_a = unittest::random_integers<U>(n);

  thrust::host_vector<U> h_keys_b   = unittest::random_integers<U>(n);
  thrust::host_vector<U> h_values_b = unittest::random_integers<U>(n);

  thrust::host_vector<T> h_a(n), h_b(n);
  for(size_t i = 0; i < n; ++i)
  {
    h_a[i] = T(h_keys_a[i], h_values_a[i]);
    h_b[i] = T(h_keys_b[i], h_values_b[i]);
  }

  thrust::stable_sort(h_a.begin(), h_a.end(), thrust::greater<T>());
  thrust::stable_sort(h_b.begin(), h_b.end(), thrust::greater<T>());

  thrust::device_vector<T> d_a = h_a;
  thrust::device_vector<T> d_b = h_b;

  thrust::host_vector<T>   h_result(h_a.size() + h_b.size());
  thrust::device_vector<T> d_result(d_a.size() + d_b.size());

  typename thrust::host_vector<T>::iterator   h_end;
  typename thrust::device_vector<T>::iterator d_end;
  
  h_end = thrust::set_union(h_a.begin(), h_a.end(),
                            h_b.begin(), h_b.end(),
                            h_result.begin(),
                            thrust::greater<T>());
  h_result.erase(h_end, h_result.end());

  d_end = thrust::set_union(d_a.begin(), d_a.end(),
                            d_b.begin(), d_b.end(),
                            d_result.begin(),
                            thrust::greater<T>());
  d_result.erase(d_end, d_result.end());

  ASSERT_EQUAL_QUIET(h_result, d_result);
}
DECLARE_VARIABLE_UNITTEST(TestSetUnionKeyValueDescending);

