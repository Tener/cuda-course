#include <unittest/unittest.h>
#include <thrust/replace.h>

template <class Vector>
void TestReplaceSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  2; 
    data[2] =  1;
    data[3] =  3; 
    data[4] =  2; 

    thrust::replace(data.begin(), data.end(), (T) 1, (T) 4);
    thrust::replace(data.begin(), data.end(), (T) 2, (T) 5);

    Vector result(5);
    result[0] =  4; 
    result[1] =  5; 
    result[2] =  4;
    result[3] =  3; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceSimple);


template <typename T>
void TestReplace(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    T old_value = 0;
    T new_value = 1;

    thrust::replace(h_data.begin(), h_data.end(), old_value, new_value);
    thrust::replace(d_data.begin(), d_data.end(), old_value, new_value);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplace);


template <class Vector>
void TestReplaceCopySimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] = 1; 
    data[1] = 2; 
    data[2] = 1;
    data[3] = 3; 
    data[4] = 2; 

    Vector dest(5);

    thrust::replace_copy(data.begin(), data.end(), dest.begin(), (T) 1, (T) 4);
    thrust::replace_copy(dest.begin(), dest.end(), dest.begin(), (T) 2, (T) 5);

    Vector result(5);
    result[0] = 4; 
    result[1] = 5; 
    result[2] = 4;
    result[3] = 3; 
    result[4] = 5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopySimple);


template <typename T>
void TestReplaceCopy(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;
    
    T old_value = 0;
    T new_value = 1;
    
    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy(h_data.begin(), h_data.end(), h_dest.begin(), old_value, new_value);
    thrust::replace_copy(d_data.begin(), d_data.end(), d_dest.begin(), old_value, new_value);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopy);



template <typename T>
struct less_than_five
{
  __host__ __device__ bool operator()(const T &val) const {return val < 5;}
};

template <class Vector>
void TestReplaceIfSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    thrust::replace_if(data.begin(), data.end(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  0; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceIfSimple);


template <class Vector>
void TestReplaceIfStencilSimple(void)
{
    typedef typename Vector::value_type T;

    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector stencil(5);
    stencil[0] = 5;
    stencil[1] = 4;
    stencil[2] = 6;
    stencil[3] = 3;
    stencil[4] = 7;

    thrust::replace_if(data.begin(), data.end(), stencil.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  1; 
    result[1] =  0; 
    result[2] =  4;
    result[3] =  0; 
    result[4] =  5; 

    ASSERT_EQUAL(data, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceIfStencilSimple);


template <typename T>
void TestReplaceIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::replace_if(h_data.begin(), h_data.end(), less_than_five<T>(), (T) 0);
    thrust::replace_if(d_data.begin(), d_data.end(), less_than_five<T>(), (T) 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceIf);


template <typename T>
void TestReplaceIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_stencil = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::replace_if(h_data.begin(), h_data.end(), h_stencil.begin(), less_than_five<T>(), (T) 0);
    thrust::replace_if(d_data.begin(), d_data.end(), d_stencil.begin(), less_than_five<T>(), (T) 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceIfStencil);


template <class Vector>
void TestReplaceCopyIfSimple(void)
{
    typedef typename Vector::value_type T;
    
    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector dest(5);

    thrust::replace_copy_if(data.begin(), data.end(), dest.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  0; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopyIfSimple);


template <class Vector>
void TestReplaceCopyIfStencilSimple(void)
{
    typedef typename Vector::value_type T;
    
    Vector data(5);
    data[0] =  1; 
    data[1] =  3; 
    data[2] =  4;
    data[3] =  6; 
    data[4] =  5; 

    Vector stencil(5);
    stencil[0] = 1;
    stencil[1] = 5;
    stencil[2] = 4;
    stencil[3] = 7;
    stencil[4] = 8;

    Vector dest(5);

    thrust::replace_copy_if(data.begin(), data.end(), stencil.begin(), dest.begin(), less_than_five<T>(), (T) 0);

    Vector result(5);
    result[0] =  0; 
    result[1] =  3; 
    result[2] =  0;
    result[3] =  6; 
    result[4] =  5; 

    ASSERT_EQUAL(dest, result);
}
DECLARE_VECTOR_UNITTEST(TestReplaceCopyIfStencilSimple);


template <typename T>
void TestReplaceCopyIf(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy_if(h_data.begin(), h_data.end(), h_dest.begin(), less_than_five<T>(), 0);
    thrust::replace_copy_if(d_data.begin(), d_data.end(), d_dest.begin(), less_than_five<T>(), 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIf);

template <typename T>
void TestReplaceCopyIfStencil(const size_t n)
{
    thrust::host_vector<T>   h_data = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_data = h_data;

    thrust::host_vector<T>   h_stencil = unittest::random_samples<T>(n);
    thrust::device_vector<T> d_stencil = h_stencil;

    thrust::host_vector<T>   h_dest(n);
    thrust::device_vector<T> d_dest(n);

    thrust::replace_copy_if(h_data.begin(), h_data.end(), h_stencil.begin(), h_dest.begin(), less_than_five<T>(), 0);
    thrust::replace_copy_if(d_data.begin(), d_data.end(), d_stencil.begin(), d_dest.begin(), less_than_five<T>(), 0);

    ASSERT_ALMOST_EQUAL(h_data, d_data);
    ASSERT_ALMOST_EQUAL(h_dest, d_dest);
}
DECLARE_VARIABLE_UNITTEST(TestReplaceCopyIfStencil);

