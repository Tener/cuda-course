#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>

#include <sys/types.h>
#include <sys/stat.h>

// CUDA
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hiprand/hiprand_kernel.h>

#include <math_functions.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_reference.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

//png++
#include <png++/png.hpp>


// local includes
#define CHMUTOV_DEGREE 16

#include "constant_vars.hpp"

#include "common.hpp"
#include "graphics.hpp"

#include "polynomial.hpp"
#include "utils.hpp"
#include "chebyshev.hpp"

#include "colors.hpp"
#include "sign_change.hpp"
#include "surface.hpp"

#include "linear_algebra.hpp"


    
// Nice intro to ray tracing:
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter0.htm

//// PerspectiveRay class; base_vector specifies how we should locate ourselves in the space
//template < typename dom3 = float3, typename dom = float >
//struct PerspectiveRay
//{
//  int w;
//  int h;
//  int ix_w;
//  int ix_h;
//  dom3 direction_vector;
//  dom3 view_angle;
//  dom3 starting_point;
// 
//  dom3 current_point;
//  
//  __host__ __device__
//  PerspectiveRay( int w, int h, int ix_w, int ix_h, dom3 view_angle, dom3 starting_point, dom scale, dom3 base_vector = make_float3(1,0,0) )
//    : w(w), h(h), ix_w(ix_w), ix_h(ix_h), view_angle(view_angle), starting_point(starting_point), 
//      // XXX: direction vector should depend on w/h/ix_w/ix_h 
//      direction_vector(rotate_vector< dom3, dom >(base_vector, view_angle)) 
//  {
//  }
// 
//  // transform the current point along the ray by given distance. 
//  // negative distance reverses direction of movement.
//  __host__ __device__
//  void move_point( const dom & length )
//  {
//    current_point.x += direction_vector.x * length;
//    current_point.y += direction_vector.y * length;
//    current_point.z += direction_vector.z * length;
//  }
//};
// 
//// OrtographicRay class; base_vector specifies how we should locate ourselves in the space
//template < typename dom3 = float3, typename dom = float >
//struct OrtographicRay
//{
//  int w;
//  int h;
//  int ix_w;
//  int ix_h;
//  dom3 direction_vector;
//  dom3 view_angle;
// 
//  dom3 current_point;
// 
//  __host__ __device__
//  inline
//  dom rescale(const dom dim, const dom dim_max, const dom scale) { return scale * ((dim - (dim_max/2))/dim_max); }
//  
//  __host__
//  OrtographicRay( int w, int h, int ix_w, int ix_h, dom3 view_angle, dom3 starting_point, dom scale, dom3 base_vector = make_float3(1,0,0) )
//    : w(w), h(h), ix_w(ix_w), ix_h(ix_h), view_angle(view_angle),
//      direction_vector(rotate_vector< dom3, dom >(base_vector, view_angle)),
//      // XXX: starting point should depend on w/h/ix_w/ix_h AND direction vector
//      current_point(translate_point(starting_point, make_float3(0, rescale(ix_w,w,scale), rescale(ix_h,h,scale))))
//      //      current_point(translate_point(starting_point, rotate_vector< dom3, dom >(make_float3(0, rescale(ix_w,w,scale), rescale(ix_h,h,scale)), view_angle)))
//  {
//  }
// 
//  // transform the current point along the ray by given distance. 
//  // negative distance reverses direction of movement.
//  __host__ __device__
//  void move_point( const dom & length )
//  {
//    current_point.x += direction_vector.x * length;
//    current_point.y += direction_vector.y * length;
//    current_point.z += direction_vector.z * length;
//  }
//};

template < typename dom3 = float3, typename dom = float >
struct ModelViewRay
{
  
  dom3 current_point;
  const dom3 direction_vector;

  __host__ __device__
  inline
  dom rescale(const dom dim, const dom dim_max, const dom scale) { return scale * ((dim - (dim_max/2))/dim_max); }

  __device__ __host__
  inline
  dom3 modelview_matrix_transform( const dom3 vec )
  {
    //device_modelview_matrix[16]

    dom arrIn[4] = { vec.x, vec.y, vec.z, 1 }; // homo
    dom arrOut[4];

    for(int i = 0; i < 4; i++)
      {
        dom v = 0;
        for(int j = 0; j < 4; j++)
          {
#if __CUDA_ARCH__ > 0
            v += device_modelview_matrix[4*j + i] * arrIn[j];
#else
            v += host_modelview_matrix[4*j + i] * arrIn[j];
#endif
          }
        arrOut[i] = v;
      }

    return make_float3( arrOut[0] * arrOut[3], 
                        arrOut[1] * arrOut[3], 
                        arrOut[2] * arrOut[3] );
  } 

  __device__ __host__
  ModelViewRay( int w, int h, int ix_w, int ix_h, dom scale ) :
    current_point( modelview_matrix_transform( make_float3(rescale(ix_w,w,scale), rescale(ix_h,h,scale), 0) ) ),
    direction_vector( Normalize( modelview_matrix_transform( make_float3(0, 0, 1) ) ) )
  {
  }

  // transform the current point along the ray by given distance. 
  // negative distance reverses direction of movement.
  __device__ __host__
  inline
  void move_point( const dom & length )
  {
    current_point.x += direction_vector.x * length;
    current_point.y += direction_vector.y * length;
    current_point.z += direction_vector.z * length;
  }

};

template < typename SurfType, typename RayType >
struct RayTrace
{
  uint background;
  SurfType surface;

  int steps;
  int bisect_count;

  int w; int h; int ix_h;
  float3 view_angle; float3 starting_point; float scale;
  float view_distance; // how far do we look

  RayTrace(int w, int h, int ix_h,
           float3 view_angle, float3 starting_point, float scale,
           float view_distance, 
           int steps,
           int bisect_count,
           SurfType surface = SurfType())
  :
    w(w), h(h), ix_h(ix_h),
    view_angle(view_angle), starting_point(starting_point), scale(scale),
    view_distance(view_distance),
    steps(steps), bisect_count(bisect_count),
    background(0)
  {
  }


  __device__
  Color operator()( int ix_w )
  {
//    RayType ray( w, h, ix_w, ix_h, // which pixel on screen are we calculating
//        	 view_angle,	   // where do we look
//        	 starting_point,   // where do we start
//        	 scale		   // defines the length of '1.0' in pixels
//        	 );

    RayType ray( w, h, ix_w, ix_h, 1 );

    float surf_value = surface.calculate( ray.current_point );
    bool sign_change = false;
    float step = view_distance / steps;
    float pos = 0; // position along the ray

    // root detection 
    for(; pos < view_distance && !sign_change;)
      {
        step += (view_distance / steps) / 10; // if there is no root we go faster each step
        ray.move_point(step);
        pos += step;
        float tmp = surface.calculate( ray.current_point );
        sign_change = SignChange<>::check( surf_value, tmp );
        surf_value = tmp;
      }



    if ( sign_change )
      {
        // root refinement
        for(int i = 0; i < bisect_count; i++)
	 {
           step /= 2;
	   if ( sign_change )
	     {
	       step *= -1; // we reverse movement direction if there was a sign change
	     }

           ray.move_point(step);
           float tmp = surface.calculate( ray.current_point );
           sign_change = SignChange<>::check( surf_value, tmp );
           surf_value = tmp;
	 }

        // shade calculation
        return surface.lightning(ray.current_point, make_float3(1, 0, 0));
      }
    else
      {
        return background;
      }  
  }
};

// 
//template < typename SurfType >
//struct TracePoint
//{
//  const uint background;
//  int steps;
//  int bisect_count;
//  SurfType surfaceInstance;
// 
//  // all these variables are related to the viewport:
//  // - this will likely stay:
//  int w; int h; int ix_h; 
//  float3 R0;
//  // - but not these:
//  float3 Rd;
//  float3 Rtrans;
//  float range_w; // bounding box
//  float range_h; // bounding box
// 
//  // Vmin, Vmax, Vdiff;
//  float3 Vmin, Vmax, Vdiff;
//  float step_size;
//  
//  TracePoint(int w, int h, 
//             int ix_h, 
//             View v,
//             SurfType surfInst = SurfType())
//    : w(w), h(h), ix_h(ix_h), 
//      steps(v.steps),
//      bisect_count(v.bisect_count),
//      R0(v.starting_point),
//      Rd(v.DirectionVector),
//      step_size(sqrt(pow(R0.x - Rd.x,2) + 
//        	     pow(R0.y - Rd.y,2) + 
//        	     pow(R0.z - Rd.z,2)) / steps),
//      Rtrans(make_float3( R0.x - Rd.x, R0.y - Rd.y, R0.z - Rd.z )),
//      background(0),
//      surfaceInstance(surfInst)
//  { 
//    range_w = sqrt(pow(R0.x - Rd.x,2) + 
//                   pow(R0.y - Rd.y,2) + 
//                   pow(R0.z - Rd.z,2)) / 2;
//    range_h = range_w;
// 
//    Vmin.x = R0.x - range_w;
//    Vmin.y = R0.y - range_h;
//    Vmin.z = R0.z;
// 
//    Vmax.x = Rd.x + range_w;
//    Vmax.y = Rd.y + range_h;
//    Vmax.z = Rd.z;
// 
//    Vdiff.x = Vmax.x - Vmin.x;
//    Vdiff.y = Vmax.y - Vmin.y;
//    Vdiff.z = Vmax.z - Vmin.z;
//  };
// 
//  __host__ __device__
//  inline
//  void Ray( float3 & Rc, const float3 & Rd, const float & t )
//  {
//    Rc.x += Rd.x * t;
//    Rc.y += Rd.y * t;
//    Rc.z += Rd.z * t;
//  };
// 
//  __host__ __device__ 
//  uint operator()( int ix_w )
//  {
//    float x = 2.0f * (float)range_w * (((float)ix_w - (w/2.0f)) / (float)w);
//    float y = 2.0f * (float)range_h * (((float)ix_h - (h/2.0f)) / (float)w);
//    float z = 0 ;
// 
//    float3 Rc = make_float3( R0.x + x, R0.y + y, R0.z + z ); // current point
//    
//    float val = surfaceInstance.calculate( Rc );
//    bool sign_has_changed = false;
//    
//    for(int i = 0; (i < steps) && !sign_has_changed; i++)
//      {
// 	// calculate next position
//        Ray( Rc, Rtrans, step_size );
// 	// determine the sign
// 	float tmp = surfaceInstance.calculate(Rc);
//        sign_has_changed = SignChange<>::check( val, tmp );
// 	val = tmp;
//      }
// 
//      if ( sign_has_changed )
//     {
//       float step_size_l = step_size;
//#pragma unroll
//       for(int i = 0; i < bisect_count && i < 11; i++)
//         {
//           step_size_l /= 2 * (1 + (sign_has_changed * -2 ));
//// 	   if ( sign_has_changed )
//// 	     {
//// 	       step_size *= -1; // if there was a sign change, we swap directions
//// 	     }
//           //
//           Ray( Rc, Rtrans, step_size_l );
//           float tmp = surfaceInstance.calculate(Rc);
//           sign_has_changed = SignChange<>::check( val, tmp ); //SignChange( val, tmp );
//           val = tmp;
//         }
//       return surfaceInstance.lightning(Rc, make_float3( 1, 0, 0 ));
// 
//     }
//      else
//        {
//          return background;
//        }
//  }
//};
// 
//template < typename SurfType >
//struct TraceScreen
//{
//  static
//  void run(int w, int h, View view, uint * pbo, SurfType s = SurfType())
//  {
//    for(int ix_h = 0; ix_h < h; ix_h++)
//    {
//      thrust::transform( thrust::make_counting_iterator< short >(0),
//                         thrust::make_counting_iterator< short >(w),
//                         thrust::device_ptr< uint >(pbo + h * ix_h),
//                         TracePoint< SurfType >(w,h,ix_h,view,s));
//    }
//  }
//};





template < typename SurfType, typename RayType = ModelViewRay< > >
struct RayTraceScreen
{
  int w;
  int h;
  View view;
  uint * pbo;

    
  RayTraceScreen(int w, int h, View view, uint * pbo) :
    w(w), h(h), view(view), pbo(pbo)
  {
  }

  static
  __host__
  png::rgba_pixel
  unpack_Color(const uint rgba)
  {
    char * rgba_arr = (char *)(&rgba);
    return png::rgba_pixel( rgba_arr[0], rgba_arr[1], rgba_arr[2], 255 );
  }

  __host__
  void screenshot(const std::string filename)
  {
    png::image< png::rgba_pixel > img(w,h);
    thrust::device_ptr< uint > dev_pbo(pbo);
    thrust::host_vector< uint > pixels(dev_pbo, dev_pbo + w * h );

    thrust::host_vector< uint >::iterator pix(pixels.begin());
        
    for(int i = 0; i < img.get_width(); i++)
      for(int j = 0; j < img.get_height(); j++)
        {
          img[i][j] = unpack_Color(*pix);
          pix++;
        }
    img.write(filename);
    printf("file written! %s\n", filename.c_str());
  }

  void run()
  {
    for(int ix_h = 0; ix_h < h; ix_h++)
    {
      thrust::transform( thrust::make_counting_iterator< short >(0),
                         thrust::make_counting_iterator< short >(w),
                         thrust::device_ptr< uint >(pbo + h * ix_h),
                         RayTrace< SurfType, RayType >(w,h,ix_h,
                                                       view.angle, view.starting_point,
                                                       view.scale, view.distance, 
                                                       view.steps, view.bisect_count));
    }


    if ( view.screenshot )
      {
        char filename[256];
        sprintf(filename, "screenshots/shot_%d.png", time(0)); // XXX: make '/' portable
        screenshot(std::string(filename));
        
        // we made the screenshot. now, to not confuse 'movie' making part below, disable screenshot flag.
        view.screenshot = false;
      }

    {
      static int session_start = 0;
      static int count = 0;
      static char path[1024];
      static View last_view;

      if (!session_start)
        {
          session_start = time(0);
          sprintf(path, "movie/%s/%d", SurfString(SurfType::surface_id).c_str(), session_start);
          // call 'mkdir -p' for recursive mkdir
          {
            char cmd[1024];
            sprintf(cmd, "mkdir -p '%s'", path);
            system(cmd);
            //mkdir(path, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
          }
          last_view = view;
        }
      
      if (memcmp( (&last_view), (&view), sizeof(View)))
        {
          char filename[1024];
          sprintf(filename,"%s/%010d.png", path, count);
          count++;
          screenshot(std::string(filename));
          last_view = view;
        }
    }

  }
};

//template < typename RayType >
//struct RayDebug
//{  
//  int steps;
//  int bisect_count;
// 
//  int w; int h; int ix_h;
//  float3 view_angle; float3 starting_point; float scale;
//  float view_distance; // how far do we look
// 
//  float4 * vbo;
// 
//  RayDebug(int w, int h, int ix_h,
//           float3 view_angle, float3 starting_point, float scale,
//           float view_distance, 
//           int steps,
//           int bisect_count,
//           float4 * vbo)
//  :
//    w(w), h(h), ix_h(ix_h),
//    view_angle(view_angle), starting_point(starting_point), scale(scale),
//    view_distance(view_distance),
//    steps(steps), bisect_count(bisect_count),
//    vbo(vbo)
//  {
//  }
// 
//  __device__ __host__
//  void operator()( int ix_w )
//  {
//    //    RayType ray( w, h, ix_w, ix_h, scale );
//    RayType ray( w, h, ix_w, ix_h, // which pixel on screen are we calculating
//                 view_angle,       // where do we look
//                 starting_point,   // where do we start
//                 scale             // defines the length of '1.0' in pixels
//                 );
// 
//    float step = view_distance / steps;
// 
//    vbo += steps * ix_w;
// 
//    for(int i = 0; i < steps; i++)
//      {
//        ray.move_point(step);
//        *vbo = make_float4( ray.current_point.x, ray.current_point.y, ray.current_point.z, 1.0 );
//        vbo++;
//      }
//  }
//};
// 
//template < typename RayType = ModelViewRay< > >
//struct DebugRayTraceScreen
//{
//  static
//  void run(int w, int h, View view, float4 * vbo, uint * draw_cnt)
//  {
//    view.steps = MIN(view.steps, MAX_DEBUG_STEPS);
// 
//    for(int ix_h = 0; ix_h < h; ix_h++)
//    {
//      thrust::for_each( thrust::make_counting_iterator< short >(0),
//                        thrust::make_counting_iterator< short >(w),
//                        RayDebug< OrtographicRay< > >(w,h,ix_h,
//                                            view.angle, view.starting_point,
//                                            view.scale, view.distance, 
//                                            view.steps, view.bisect_count,
//                                            vbo + view.steps * ix_h * w) );
//    }
//    *draw_cnt = view.steps * h * w;
//  }
//};

void initModelViewMatrix(View view)
{
  PrintView( view );

  GLfloat modelViewMatrix[16];
  
  glMatrixMode(GL_MODELVIEW);
  glPushMatrix();
  {
    glLoadIdentity();
    glOrtho( -1, 1, -1, 1, -1, 1 );

    glScalef( view.scale, view.scale, view.scale ); // scale along z axis

    glRotatef( view.angle.x * 10, 1, 0, 0 );
    glRotatef( view.angle.y * 10, 0, 1, 0 );
    glRotatef( view.angle.z * 10, 0, 0, 1 );

    glTranslatef( view.starting_point.x/view.scale, 
                  view.starting_point.y/view.scale, 
                  view.starting_point.z/view.scale );

    glGetFloatv(GL_MODELVIEW_MATRIX, modelViewMatrix);
  }
  glPopMatrix();

  for(int i = 0; i < 4; i++)
    {
      printf("\t");
      for(int j = 0; j < 4; j++)
        {
          printf("%f\t", modelViewMatrix[i+j*4]);
          host_modelview_matrix[i+j*4] = modelViewMatrix[i+j*4];
        }
      printf("\n");
    }

  hipMemcpyToSymbol(HIP_SYMBOL(device_modelview_matrix), host_modelview_matrix, sizeof(float) * 16);
}

//extern "C" void launch_debug_kernel(float4 * vbo, unsigned int * draw_cnt, View view, int w, int h)
//{
//  *draw_cnt = 0;
//  initModelViewMatrix(view);
//  DebugRayTraceScreen< >::run( w, h, view, vbo, draw_cnt );
// 
//}


extern "C" void launch_raytrace_kernel(uint * pbo, View view, int w, int h)
{
//  std::cerr << "w=" << w << std::endl
//            << "h=" << h << std::endl; 

  // modelview matrix fun
  initModelViewMatrix(view);
  


#define TraceEngine RayTraceScreen

  switch ( view.surf )
    {
    case SURF_BARTH:
      TraceEngine< Surface< SURF_BARTH > >(w,h,view,pbo).run();
      break;
    case SURF_CHMUTOV:
      TraceEngine< Surface< SURF_CHMUTOV > >(w,h,view,pbo).run();
      break;
    case SURF_CHMUTOV_ALT:
      TraceEngine< Surface< SURF_CHMUTOV_ALT > >(w,h,view,pbo).run();
      break;
    case SURF_HEART:
      TraceEngine< Surface< SURF_HEART > >(w,h,view,pbo).run();
      break;
    case SURF_PLANE:
      TraceEngine< Surface< SURF_PLANE > >(w,h,view,pbo).run();
      break;
    case SURF_TORUS:
      TraceEngine< Surface< SURF_TORUS > >(w,h,view,pbo).run();
      break;
    case SURF_DING_DONG:
      TraceEngine< Surface< SURF_DING_DONG > >(w,h,view,pbo).run();
      break;
    case SURF_CAYLEY:
      TraceEngine< Surface< SURF_CAYLEY > >(w,h,view,pbo).run();
      break;
    case SURF_DIAMOND:
      TraceEngine< Surface< SURF_DIAMOND > >(w,h,view,pbo).run();
      break;
    case SURF_BALL:
      TraceEngine< Surface< SURF_BALL > >(w,h,view,pbo).run();
      break;
    case SURF_ARB_POLY:
      {
        // i'm tired of doing this the clean way... so let's just make a hack.
        // copy arbitrary polynomial's parameters
        for(int i = 0; i < 3; i++)
          {
            size_t stride = sizeof(float) * (18+1);
            Polynomial<> p(view.arb_poly[i]);
            hipMemcpyToSymbol(HIP_SYMBOL( arb_poly_const_coeff), p.coeff, stride, stride * i );
            hipMemcpyToSymbol(HIP_SYMBOL( arb_poly_const_coeff_der), p.coeff_der, stride, stride * i );
            stride = sizeof(int);
            hipMemcpyToSymbol(HIP_SYMBOL( arb_poly_const_size), &p.max_deg, stride, stride * i );
          }
        TraceEngine< Surface< SURF_ARB_POLY > >(w,h,view,pbo).run();
        break;
      }
    default:
      break;
    }

#undef TraceEngine
}
