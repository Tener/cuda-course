#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>

// CUDA
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hiprand/hiprand_kernel.h>

#include <math_functions.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_reference.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "common.hpp"
#include "graphics.hpp"

#include "utils.hpp"

#define CHMUTOV_DEGREE 16

__host__ __device__
inline uint RGBA( unsigned char r, unsigned char g, unsigned char b, unsigned char a )
{ 
  return 
    (a << (3 * 8)) + 
    (b << (2 * 8)) +
    (g << (1 * 8)) +
    (r << (0 * 8));
}

__device__ __host__
float Chebyshev_Pol_N( int N, float x )
{
  float arr[CHMUTOV_DEGREE+1];
  //  thrust::device_vector< float > arr( N );
  arr[0] = 1;
  arr[1] = x;
#pragma unroll 16
  for(unsigned int i = 2; i < N+1; i++)
    {
      arr[i] = 2 * x * arr[i-1] - arr[i-2];
    }
  return arr[N];
}

template <int N>
struct Chebyshev_DiVar
{
  __host__ __device__
  static float calculate(float x)
  {
    float arr_0 = 1;
    float arr_1 = x;
#pragma unroll 16
    for(unsigned int i = 1; i < N/2; i++)
      {
        arr_0 = 2 * x * arr_0 - arr_1;
        arr_1 = 2 * x * arr_1 - arr_0;
      }
    return arr_0;
  }
};


template <int N>
struct Chebyshev_Pol
{
  __host__ __device__
  static float calculate(float x)
  {
    float arr[N+1];
    arr[0] = 1;
    arr[1] = x;
#pragma unroll 16
    for(unsigned int i = 2; i < N+1; i++)
      {
	arr[i] = 2 * x * arr[i-1] - arr[i-2];
      }
    return arr[N];
  }
};

template <int N>
struct Chebyshev_T
{
  __host__ __device__
  static float calculate(float x)
  { 
    return 2 * x * Chebyshev_T< N-1 >::calculate(x) - Chebyshev_T< N-2 >::calculate(x);
  };
};

template <>
struct Chebyshev_T< 0 >
{
  __host__ __device__
  static float calculate(float x)
  { 
    return 1;
  };
};

template <>
struct Chebyshev_T< 1 >
{
  __host__ __device__
  static float calculate(float x)
  { 
    return x;
  };
};


//inline 
//__host__ __device__ 
//float Chebyshev_Pol( int N, float x )
//{
//  
//  if ( N == 0 )
//    return 1;
//  if ( N == 1 )
//    return x;
//  return 2 * x * Chebyshev_Pol(N-1, x) - Chebyshev_Pol(N-2, x);
//}


    
// Nice intro to ray tracing:
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter0.htm

struct TracePoint
{
  int w; int h; int ix_h;
  Surf surf;
  int steps;
  int bisect_count;
  
  float3 R0;
  float3 Rd;
  
  float3 Rtrans;

  // bounding box
  float range_w;
  float range_h;

  // Vmin, Vmax, Vdiff;
  float3 Vmin, Vmax, Vdiff;

  float step_size;
  
  TracePoint(int w, int h, 
             int ix_h, 
             View v)
    : w(w), h(h), ix_h(ix_h), 
      surf(v.surf),
      steps(v.steps),
      bisect_count(v.bisect_count),
      R0(v.StartingPoint),
      Rd(v.DirectionVector),
      range_w(v.range_w),
      range_h(v.range_h),
      step_size(sqrt(pow(R0.x - Rd.x,2) + 
		     pow(R0.y - Rd.y,2) + 
		     pow(R0.z - Rd.z,2)) / steps),
      Rtrans(make_float3( R0.x - Rd.x, R0.y - Rd.y, R0.z - Rd.z ))
  { 
    Vmin.x = R0.x - range_w;
    Vmin.y = R0.y - range_h;
    Vmin.z = R0.z;

    Vmax.x = Rd.x + range_w;
    Vmax.y = Rd.y + range_h;
    Vmax.z = Rd.z;

    Vdiff.x = Vmax.x - Vmin.x;
    Vdiff.y = Vmax.y - Vmin.y;
    Vdiff.z = Vmax.z - Vmin.z;
  };

  __host__ __device__
  inline
  float Chebyshev( char n, float x )
  { // http://en.wikipedia.org/wiki/Chebyshev_polynomials
    return 
      ( x <= -1 ) ? ((n & 1 ? -1 : 1) * cosh( n * acosh( -x ) )) :
      (( x >= 1 ) ? cosh( n * acosh( x ) ) : cos(n * acos(x)));
  };

  __host__ __device__
  inline
  float Surface(float3 V,enum Surf surf_id)
  {
    float x, y, z;
    x = V.x; y = V.y; z = V.z;

    switch ( surf_id )
      {
      case SURF_DING_DONG:
        {
          return x*x+y*y-z*(1-z*z);
        }

      case SURF_CHMUTOV_0:
	return Chebyshev( CHMUTOV_DEGREE, V.x ) + 
	       Chebyshev( CHMUTOV_DEGREE, V.y ) + 
	       Chebyshev( CHMUTOV_DEGREE, V.z );

      case SURF_CHMUTOV_1:
	return Chebyshev_T< CHMUTOV_DEGREE >::calculate( V.x ) + 
	       Chebyshev_T< CHMUTOV_DEGREE >::calculate( V.y ) + 
	       Chebyshev_T< CHMUTOV_DEGREE >::calculate( V.z );

      case SURF_CHMUTOV_2:
	return Chebyshev_Pol< CHMUTOV_DEGREE >::calculate(V.x)
             + Chebyshev_Pol< CHMUTOV_DEGREE >::calculate(V.y)
             + Chebyshev_Pol< CHMUTOV_DEGREE >::calculate(V.z);

      case SURF_CHMUTOV_3:
	return Chebyshev_DiVar< CHMUTOV_DEGREE >::calculate(V.x)
	     + Chebyshev_DiVar< CHMUTOV_DEGREE >::calculate(V.y)
	     + Chebyshev_DiVar< CHMUTOV_DEGREE >::calculate(V.z);

//        return Chebyshev_Pol_N( CHMUTOV_DEGREE, V.x)
//             + Chebyshev_Pol_N( CHMUTOV_DEGREE, V.y)
//             + Chebyshev_Pol_N( CHMUTOV_DEGREE, V.z);

      case SURF_TORUS:
        {
          float c = .3;
          float a = 3;
          return pow(c - x*x + y*y, 2 ) + z*z - a*a;
        }
      case SURF_DIAMOND:
        {
          return sin(x) * sin(y) * sin(z) + sin(x) * cos(y) * cos(z) + cos(x) * sin(y) * cos(z) + cos(x) * cos(y) * sin(z);
        }
      case SURF_BALL:
        {
          return sqrt(x * x + y * y + z * z) - 1;
        }
      case SURF_CAYLEY:
        {
          return -5 * (x * x * (y + z) + y * y * (x + z) + z * z * (x + y)) + 2 * (x * y + y * x + x * z);
        }
      }

    return 0;
        
  };

  __host__ __device__
  inline
  void Ray( float3 & Rc, const float3 & Rd, const float & t )
  {
    Rc.x += Rd.x * t;
    Rc.y += Rd.y * t;
    Rc.z += Rd.z * t;
  };

  __host__ __device__
  inline
  // this is likely to be slow
  bool SignChangeSlow( const float & a, const float & b )
  {
    if ( a < 0 ) // a is below 0
      {
	return !(b < 0);
      }
    else 
      if (a > 0) // a is above 0
	{
	  return !(b > 0);
	}
      else // a is equal to 0
	{
	  return (b != 0);
	}
  };

  __host__ __device__
  bool SignChangeBit( const float & a, const float & b )
  {
    return signbit(a) != signbit(b);
  }

  __host__ __device__
  bool SignChange( const float & a, const float & b )
  {
    /*
0      a < 0
1      a > 0
2      0 ^ 1

3      b < 0
4      b > 0
5      3 ^ 4

     (0 ^ 3)
  || (1 ^ 4)
  || (2 ^ 5)
  
    */
    
    bool d0 = a < 0;
    bool d1 = a > 0;
    bool d2 = d0 ^ d1;
    bool d3 = b < 0;
    bool d4 = b > 0;
    bool d5 = d3 ^ d4;
    
    return (d0 ^ d3) || (d1 ^ d4) || (d2 ^ d5);
  }

  __host__ __device__
  void Normalize( float3 & Vec )
  {
    float len = sqrt(Vec.x * Vec.x + Vec.y * Vec.y + Vec.z * Vec.z);
    Vec.x /= len;
    Vec.y /= len;
    Vec.z /= len;
  }

  __host__ __device__ 
  uint operator()( int ix_w )
  {
    float x = 2.0f * (float)range_w * (((float)ix_w - (w/2.0f)) / (float)w);
    float y = 2.0f * (float)range_h * (((float)ix_h - (h/2.0f)) / (float)w);
    float z = 0 ;

    float3 Rc = make_float3( R0.x + x, R0.y + y, R0.z + z ); // current point
    
    float val = Surface( Rc, surf );
    bool sign_has_changed = false;
    
    for(int i = 0; (i < steps) && !sign_has_changed; i++)
      {
 	// calculate next position
        Ray( Rc, Rtrans, step_size );
 	// determine the sign
 	float tmp = Surface(Rc,surf);
	sign_has_changed = SignChangeSlow( val, tmp ); //SignChange( val, tmp );
 	val = tmp;
      }

      if ( sign_has_changed )
     {
       float step_size_l = step_size;
#pragma unroll 10
       for(int i = 0; i < bisect_count; i++)
	 {
	   step_size_l /= sign_has_changed ? -2 : 2;
// 	   if ( sign_has_changed )
// 	     {
// 	       step_size *= -1; // if there was a sign change, we swap directions
// 	     }
	   //
	   Ray( Rc, Rtrans, step_size_l );
	   float tmp = Surface(Rc,surf);
	   sign_has_changed = SignChangeSlow( val, tmp ); //SignChange( val, tmp );
	   val = tmp;
	 }

#define EXPDAMP( p ) (10.0f + 240.0f * (expf(-fabsf(p))))
       return RGBA( EXPDAMP( Rc.x ),
                    EXPDAMP( Rc.y ),
                    EXPDAMP( Rc.z ),
                    0);


//#define COLOR( p, pmin, pmax ) (10.0f + 240.0f * fabs((p-pmin)/(pmax-pmin)) )
// 
//       return RGBA( COLOR( Rc.x, Vmin.x, Vmax.x ),
//        	    COLOR( Rc.y, Vmin.y, Vmax.y ),
//        	    COLOR( Rc.z, Vmin.z, Vmax.z ),
//        	    0);
// 
//#undef COLOR
 

//#define TRANS( x ) fabs(240 * (x + 1) / 2)
// 
//       return RGBA( TRANS(Rc.x) + 10, 
// 		    TRANS(Rc.y) + 10,
// 		    TRANS(Rc.z) + 10,
//                    0); 
//#undef TRANS
       
     }
      else
        {
          return RGBA( 0, 
                       0,
                       0,  
                       0);
        }
  }
};

extern "C" void launch_raytrace_kernel(uint * pbo, View view, int w, int h)
{
  std::cerr << "w=" << w << std::endl
            << "h=" << h << std::endl; 

  PrintView( view );

  for(int ix_h = 0; ix_h < h; ix_h++)
    {
      thrust::transform( thrust::make_counting_iterator< short >(0),
                         thrust::make_counting_iterator< short >(w),
                         thrust::device_ptr< uint >(pbo + h * ix_h),
                         TracePoint(w,h,ix_h,
                                    view));
    }

}
