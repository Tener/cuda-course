#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <iostream>

// CUDA
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hiprand/hiprand_kernel.h>

#include <math_functions.h>

// thrust
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/device_reference.h>
#include <thrust/device_vector.h>
#include <thrust/for_each.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/partition.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#include "common.hpp"
#include "graphics.hpp"


#define CHMUTOV_DEGREE 16

__host__ __device__
inline uint RGBA( unsigned char r, unsigned char g, unsigned char b, unsigned char a )
{ 
  return 
    (a << (3 * 8)) + 
    (b << (2 * 8)) +
    (g << (1 * 8)) +
    (r << (0 * 8));
}


enum Surf { SURF_CHMUTOV, SURF_PLANE, SURF_TORUS, SURF_DING_DONG, SURF_CAYLEY, SURF_DIAMOND };    
    
    
// Nice intro to ray tracing:
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter0.htm

struct TracePoint
{
  int w; int h; int ix_h;

  enum Surf surf;

  float3 R0; // origin point

  TracePoint(int w, int h, 
             int ix_h, 
             float3 R0 = make_float3( -1, -1, -1 ),
             enum Surf surf = SURF_CHMUTOV)
    : w(w), h(h), ix_h(ix_h), R0(R0), surf(surf) { };

  __host__ __device__
  inline
  float Chebyshev( char n, float x )
  { // http://en.wikipedia.org/wiki/Chebyshev_polynomials
    return 
      ( x <= -1 ) ? ((n & 1 ? -1 : 1) * cosh( n * acosh( -x ) )) :
      (( x >= 1 ) ? cosh( n * acosh( x ) ) : cos(n * acos(x)));
  };

  __host__ __device__
  inline
  float Surface(float3 V,enum Surf surf_id)
  {
    float x, y, z;
    x = V.x; y = V.y; z = V.z;

    switch ( surf_id )
      {
      case SURF_CHMUTOV:
        // for now - let's choose chebyshev's polynomials
        return Chebyshev( CHMUTOV_DEGREE, V.x ) + Chebyshev( CHMUTOV_DEGREE, V.y ) + Chebyshev( CHMUTOV_DEGREE, V.z );
      case SURF_TORUS:
        {
          float c = 3;
          float a = .5;
          return pow(c - x*x + y*y, 2 ) + z*z - a*a;
        }
      case SURF_DIAMOND:
        {
          return sin(x) * sin(y) * sin(z) + sin(x) * cos(y) * cos(z) + cos(x) * sin(y) * cos(z) + cos(x) * cos(y) * sin(z);
        }
      }

    return 0;
        
  };

  __host__ __device__
  inline
  void Ray( float3 & Rc, const float3 & R0, const float3 & Rd, const float & t )
  {
    Rc.x = R0.x + Rd.x * t;
    Rc.y = R0.y + Rd.y * t;
    Rc.z = R0.z + Rd.z * t;
  };

  __host__ __device__
  inline
  // this is likely to be slow
  bool SignChangeSlow( const float & a, const float & b )
  {
    if ( a < 0 ) // a is below 0
      {
	return !(b < 0);
      }
    else 
      if (a > 0) // a is above 0
	{
	  return !(b > 0);
	}
      else // a is equal to 0
	{
	  return (b != 0);
	}
  };


  __host__ __device__
  bool SignChange( const float & a, const float & b )
  {
    /*
0      a < 0
1      a > 0
2      0 ^ 1

3      b < 0
4      b > 0
5      3 ^ 4

     (0 ^ 3)
  || (1 ^ 4)
  || (2 ^ 5)
  
    */
    
    bool d0 = a < 0;
    bool d1 = a > 0;
    bool d2 = d0 ^ d1;
    bool d3 = b > 0;
    bool d4 = b < 0;
    bool d5 = d3 ^ d4;
    
    return (d0 ^ d3) || (d1 ^ d4) || (d2 ^ d5);
  }

  __host__ __device__
  void Normalize( float3 & Vec )
  {
    float len = sqrt(Vec.x * Vec.x + Vec.y * Vec.y + Vec.z * Vec.z);
    Vec.x /= len;
    Vec.y /= len;
    Vec.z /= len;
  }

  __host__ __device__
  void PrintVector( const float3 & Vec )
  {
    printf("Vec=(%f,%f,%f)\n", Vec.x, Vec.y, Vec.z );
  }

  __host__ __device__ 
  uint operator()( int ix_w )
  {
   const int max_cnt = 500;
   const float step = 1;
   
   // directon vector
   float3 Rd = make_float3( -(float)(w/2) + ix_w, 
                            -(float)(h/2) + ix_h, 
                            1); 
   // must be normalized!
   Normalize( Rd );
   //PrintVector( Rd );

   float3 Rc; // current point
   
   float val = Surface( R0, surf ); // current surface value
   bool sign_has_changed = false;
   for(int i = 0; (i < max_cnt) && !sign_has_changed; i++)
      {
 	// calculate our current position
 	Ray( Rc, R0, Rd, i * step );
 	// determine the sign
 	float tmp = Surface(Rc,surf);
	
	sign_has_changed = SignChangeSlow( val, tmp ); //SignChange( val, tmp );
 	val = tmp;
      }
 
   if ( sign_has_changed )
     {
       //       printf("(%f,%f,%f)\n", Rc.x, Rc.y, Rc.z);

#define TRANS( x ) (240 * (x + 1) / 2)

       //       printf("%f\n%f\n%f\n", Rc.x, Rc.y, Rc.z);

       return RGBA( TRANS(Rc.x) + 10, 
                    TRANS(Rc.y) + 10,
                    TRANS(Rc.z) + 10,
                    0); 

#undef TRANS

     }
   else
     {
       return RGBA( 0, 
                    0,
                    0,  
                    0);
     }
 
  }
};


extern "C" void launch_raytrace_kernel(uint * pbo, int w, int h)
{
  std::cerr << "w=" << w << std::endl
            << "h=" << h << std::endl; 

  static float3 R = make_float3(0,0,-1);
  static float cnt = 0;

  R.x = 3 * sin(cnt);
  R.y = 3 * cos(cnt);
  
  cnt += .01;

  for(int ix_h = 0; ix_h < h; ix_h++)
    {
      thrust::transform( thrust::make_counting_iterator< short >(0),
                         thrust::make_counting_iterator< short >(w),
                         thrust::device_ptr< uint >(pbo + h * ix_h),
                         TracePoint(w,h,ix_h,R,SURF_DIAMOND) );
    }

}
