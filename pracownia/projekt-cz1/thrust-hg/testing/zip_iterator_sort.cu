#include <unittest/unittest.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>

using namespace unittest;
using namespace thrust;


template <typename T>
  struct TestZipIteratorStableSort
{
  void operator()(const size_t n)
  {
#if (THRUST_HOST_COMPILER == THRUST_HOST_COMPILER_MSVC) && (_MSC_VER == 1400) && defined(_DEBUG)
    // fails on msvc80 SP1 in debug mode
    KNOWN_FAILURE;
#else    
      thrust::host_vector<T>   h1 = unittest::random_integers<T>(n);
      thrust::host_vector<T>   h2 = unittest::random_integers<T>(n);
      
      thrust::device_vector<T> d1 = h1;
      thrust::device_vector<T> d2 = h2;
      
      // sort on host
      thrust::stable_sort( make_zip_iterator(make_tuple(h1.begin(), h2.begin())),
                           make_zip_iterator(make_tuple(h1.end(),   h2.end())) );

      // sort on device
      thrust::stable_sort( make_zip_iterator(make_tuple(d1.begin(), d2.begin())),
                           make_zip_iterator(make_tuple(d1.end(),   d2.end())) );
  
      ASSERT_EQUAL_QUIET(h1, d1);
      ASSERT_EQUAL_QUIET(h2, d2);
#endif      
  }
};
VariableUnitTest<TestZipIteratorStableSort, unittest::type_list<char,short,int> > TestZipIteratorStableSortInstance;

